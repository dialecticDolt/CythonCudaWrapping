#include "hip/hip_runtime.h"
#include<stdio.h>

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

void vector_print(float *in, int n){
    for(int i = 0; i < n; i++){
        printf("%.6f, ", in[i]);
    }
}


/* The CUDA Kernel */
__global__ void vector_add_cu(float *out, float *a, float *b, int n){
	for(int i = 0; i < n; i++){
		out[i] = a[i] + b[i];
	}
}


/* The function to be wrapped by Cython */
void addition(float *out, float *a, float *b, int N){
    
    float *d_a, *d_b, *d_out;    

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    vector_add_cu<<<1, 1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

/* Sample script to check CUDA Kernel */
int main(){
    float *a, *b, *out; 
    const int N = 100;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    addition_driver(out, a, b, N);

    vector_print(out, N);

    free(a);
    free(b);
    free(out);
}


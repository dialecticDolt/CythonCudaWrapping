#include "hip/hip_runtime.h"
#include "simple.hpp"

/*The CUDA kernel */
__global__ void vector_add_cu(float *out, float *a, float *b, int n){
	for(int i = 0; i < n; i++){
		out[i] = a[i] + b[i];
	}
}

/* Implementation of the function to be wrapped by Cython */
void addition(float *out, float *a, float *b, int N){
    
    float *d_a, *d_b, *d_out;    

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    vector_add_cu<<<1, 1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}
